
#include <hip/hip_runtime.h>
#include <iostream>
#include "hip/hip_runtime_api.h"
#include "cuda_device_runtime_api.h"
#include "stdio.h"


__global__ void pinnedMemoryKernel(float* dataPtr)
{
    printf("thread id -> %d, data -> %f\n", threadIdx.x, dataPtr[threadIdx.x]);
}


void loadHostData(float* h_ptr, int size)
{
    for(int i = 0; i < size; i++)
    {
        h_ptr[i] = i * 2;
    }
}

int main() {
    std::cout << "Hello, World!" << std::endl;

    float *h_data;
    float *d_data;
    size_t data_size_bytes = sizeof(float ) * 100;
    hipError_t err = hipHostMalloc((void **) &h_data, data_size_bytes, hipHostMallocDefault); // allocate pinned mem on host
    std::cout << hipGetErrorString(err) << std::endl;
    //
    loadHostData(h_data, 100);
    //
    hipMalloc((float **)&d_data, sizeof(float) * 100); // allocate device mem
    hipMemcpy(d_data, h_data, sizeof(float ) * 100, hipMemcpyHostToDevice); // copy pinned mem to device mem
    //
    pinnedMemoryKernel<<<1, 100>>>(d_data);
    hipDeviceSynchronize();
    //
    hipDeviceReset();
    //
    hipHostFree(h_data);
    hipFree(d_data);
    return 0;
}
