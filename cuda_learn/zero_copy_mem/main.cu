#include <iostream>
#include "hip/hip_runtime_api.h"

#include "hip/hip_runtime.h"
#include "cuda_device_runtime_api.h"

/*
 * Kernel function accesses data elements from a zero copy memory
 */
__global__ void zero_copy_kernel(float* dev_data)
{
    dev_data[threadIdx.x] *= 2;

}

void fill_host_data(float* h_data, int size)
{
    for(int i = 0; i < size; i++)
    {
        h_data[i] = i;
    }
}

void print_zero_copy_mem_data(float* mapped_mem, int size)
{
    for(int i = 0; i < size; i++)
    {
        printf("mem[%d] = %f\n", i, mapped_mem[i]);
    }
}

int main() {
    std::cout << "Hello, World!" << std::endl;
    //
    // setup
    float* h_data;
    int n_bytes = sizeof(float) * 100; // 100 float numbers

    // allocate zero-copy mem on host
    hipError_t err = hipHostAlloc((void**)&h_data, n_bytes, hipHostMallocMapped);
    printf("%s\n", hipGetErrorString(err));

    // fill data
    fill_host_data(h_data, 100);

    // print out
    print_zero_copy_mem_data(h_data, 100);

    // get a device side pointer to the mapped data
    float* dev_ptr;
    err = hipHostGetDevicePointer((void**)&dev_ptr, h_data, 0);
    printf("%s\n", hipGetErrorString(err));

    // launch kernel, NOTE: we do not need explicit memory copying now
    zero_copy_kernel<<<1, 100>>>(dev_ptr);
    hipDeviceSynchronize();

    // now print - kernel should have multiplied every entry by 2 in place
    print_zero_copy_mem_data(h_data, 100);


    hipFree(dev_ptr);
    hipHostFree(h_data);
    hipDeviceReset();



    return 0;
}
